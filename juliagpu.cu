#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "juliagpu.h"
#include "cpu_bitmap.h"


#define DIM 300

struct cuComplexgpu {
	float r;
	float i;
	__device__ cuComplexgpu (float a, float b) : r(a), i(b){}
	__device__ float magnitude2(void) { return r * r + i * i; }
	__device__ cuComplexgpu operator*(const cuComplexgpu& a) {
		return cuComplexgpu(r * a.r - i * a.i, i * a.r + r*a.i);
	}
	__device__ cuComplexgpu operator+(const cuComplexgpu& a) {
		return cuComplexgpu(r + a.r, i + a.i);
	}
};

__device__ int juliagpu (int x, int y ) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x)/(DIM / 2);
	float jy = scale * (float)(DIM/2 - y)/(DIM / 2);

	cuComplexgpu c(-0.8, 0.156);
	cuComplexgpu a(jx, jy);

	for (int i = 0; i < 200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;		
	}

	return 1;
}

__global__ void kernelgpu (unsigned char *ptr) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	int juliaValue = juliagpu(x, y);
	ptr[offset * 4 + 0] = 0;
	ptr[offset * 4 + 1] = 255 * juliaValue;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
	
}

__device__ int getIndex(int x,int y){
	return (y*DIM*4) + (x*4);
}


__global__ void blur(unsigned char *ptr){

	int x = blockIdx.x;
	int y = blockIdx.y;
	int i = 0;
	int moyenneCouleur[3] = {0,0,0};
	int cpt = 0;

	
	//on r�cup�re l'index et la couleur pour chaque block de pixel
	for (int i = -4; i<5;i++){
		for (int j = -4; j<5; j++){
			if(i+x >= 0 && j+y >= 0 && x+i < DIM && y+j < DIM){
				int index = getIndex(i+x,j+y);
				moyenneCouleur[0] += ptr[index];
				moyenneCouleur[1] += ptr[index + 1];
				moyenneCouleur[2] += ptr[index + 2];
				cpt++;
			}
		}
	}

	//synchro des threads 
	 __syncthreads();

    //on r��crit les couleurs
	int index = getIndex(x,y);
	ptr[index] = (moyenneCouleur[0] / cpt);
	ptr[index+1] = (moyenneCouleur[1] / cpt);
	ptr[index+2] = (moyenneCouleur[2] / cpt);
	ptr[index+3] = 255;
}


void main () {
	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;

	hipMalloc( (void**)&dev_bitmap, bitmap.image_size());

	dim3 grid(DIM, DIM);
	kernelgpu<<<grid, 1>>>(dev_bitmap);
	
	blur<<<grid, 1>>>(dev_bitmap);
	
	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);	

	bitmap.display_and_exit();

	hipFree(dev_bitmap);
}
